//#include "filter_cub_flagged.cuh"
#include "kernel/kernel_cub_flagged.cu"


// key value pair for extremes of the polygon
hipcub::KeyValuePair<int, float> *d_ri, *d_le, *d_lo, *d_up;
hipcub::KeyValuePair<int, float>  *d_c1, *d_c2, *d_c3, *d_c4;

filter_cub_flagged::filter_cub_flagged(float *x_in, float *y_in, INDEX size2){
    x = x_in;
    y = y_in;
    n = size2;
    cub_flagged();
    //f_cub_flagged();

    // save the time for deleting
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    //printf("size of q: %d\n", size);

    // cuda copy d_q to host
    h_q = new INDEX[size];
    hipMemcpy(h_q, d_q, sizeof(INDEX) * size, hipMemcpyDeviceToHost); kernelCallCheck();
    //print_extremes();

    // save time
    // save the time for copying to host
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_copy2host = (milliseconds - t_copy2host) / step + t_copy2host;
}

void filter_cub_flagged::cub_flagged(){
    // GET CUDA TIME
    hipDeviceSynchronize();
    step++;
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // cuda malloc for x and y arrays
    hipMalloc(&d_x, sizeof(float) * n);
    hipMalloc(&d_y, sizeof(float) * n);
    // cuda mem copy for x and y arrays
    hipMemcpy(d_x, x, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc(&d_ri, sizeof(hipcub::KeyValuePair<INDEX, float>));
    hipMalloc(&d_le, sizeof(hipcub::KeyValuePair<INDEX, float>));
    hipMalloc(&d_lo, sizeof(hipcub::KeyValuePair<INDEX, float>));
    hipMalloc(&d_up, sizeof(hipcub::KeyValuePair<INDEX, float>));

    // save the time for copying to device
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_copy2device = (milliseconds - t_copy2device) / step + t_copy2device;  


    hipEvent_t start_filter, stop_filter;
    hipEventCreate(&start_filter);
    hipEventCreate(&stop_filter);
    hipEventRecord(start_filter);

    // get the time for finding axis extreme points
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // find axis extreme points
    findMax_cub<float>(d_ri, d_x, n);
    findMax_cub<float>(d_up, d_y, n);
    findMin_cub<float>(d_le, d_x, n);
    findMin_cub<float>(d_lo, d_y, n);

    // copy from d_ri to host
    hipMemcpy(&ri, d_ri, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&up, d_up, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&le, d_le, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&lo, d_lo, sizeof(int), hipMemcpyDeviceToHost);

    xri = x[ri]; yri = y[ri];
    xle = x[le]; yle = y[le];
    xlo = x[lo]; ylo = y[lo];
    xup = x[up]; yup = y[up];

    // save the time for finding axis extreme points
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_find_extremes = (milliseconds - t_find_extremes) / step + t_find_extremes;

    // get the time for finding corner points
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    // index corner points
    hipMalloc(&d_c1, sizeof(hipcub::KeyValuePair<int, float>));
    hipMalloc(&d_c2, sizeof(hipcub::KeyValuePair<int, float>));
    hipMalloc(&d_c3, sizeof(hipcub::KeyValuePair<int, float>));
    hipMalloc(&d_c4, sizeof(hipcub::KeyValuePair<int, float>));

    hipMalloc(&d_c, sizeof(float) * n);

    // compute the manhattan distance and find the minimum
    // corner 1
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, xri, yup, n);
    hipDeviceSynchronize();
    findMin_cub<float>(d_c1, d_c, n);

    // corner 2
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, xle, yup, n);
    hipDeviceSynchronize();
    findMin_cub<float>(d_c2, d_c, n);

    // corner 3
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, xle, ylo, n);
    hipDeviceSynchronize();
    findMin_cub<float>(d_c3, d_c, n);

    // corner 4
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, xri, ylo, n);
    hipDeviceSynchronize();
    findMin_cub<float>(d_c4, d_c, n);

    // copy from d_c1 to host
    hipMemcpy(&c1, d_c1, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&c2, d_c2, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&c3, d_c3, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&c4, d_c4, sizeof(INDEX), hipMemcpyDeviceToHost);

    xc1 = x[c1]; yc1 = y[c1];
    xc2 = x[c2]; yc2 = y[c2];
    xc3 = x[c3]; yc3 = y[c3];
    xc4 = x[c4]; yc4 = y[c4];


    // save the time for finding corner points
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_find_corners = (milliseconds - t_find_corners) / step + t_find_corners;

    // get the time for finding points in q
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    computeSlopes(); 
    
    hipMalloc(&d_vec_inQ, sizeof(char) * n);
    hipMalloc(&d_qa, sizeof(INDEX) * n);
    hipMalloc(&d_q, sizeof(INDEX) * n);
    hipMalloc(&d_size, sizeof(INDEX));

    // is in polygon?
    kernel_inPointsInQ<char><<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_x, d_y, n, 
                xri, yri, xup, yup, xle, yle, xlo, ylo,
                xc1, yc1, xc2, yc2, xc3, yc3, xc4, yc4,
                m1, m2, m3, m4, mh, m1b, m2b, m3b, m4b, 
                d_vec_inQ, d_qa ); kernelCallCheck();
    hipDeviceSynchronize();

    // save the time for finding points in q
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_find_points_in_Q = (milliseconds - t_find_points_in_Q) / step + t_find_points_in_Q;

    // get the time for compacting 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    compaction_cub<INDEX,char>(d_q, d_size, d_vec_inQ, d_qa, n);
    
    hipMemcpy(&size, d_size, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // save the time for compacting
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    t_compaction = (milliseconds - t_compaction) / step + t_compaction;
    

    // save the time for compacting
    hipEventRecord(stop_filter);
    hipEventSynchronize(stop_filter);
    hipEventElapsedTime(&milliseconds, start_filter, stop_filter);    
    t_total = (milliseconds - t_total) / step + t_total;
}

void filter_cub_flagged::f_cub_flagged(){
    // cuda malloc for x and y arrays
    hipMalloc(&d_x, sizeof(float) * n);
    hipMalloc(&d_y, sizeof(float) * n);
    // cuda mem copy for x and y arrays
    hipMemcpy(d_x, x, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc(&d_ri, sizeof(hipcub::KeyValuePair<int, float>));
    hipMalloc(&d_le, sizeof(hipcub::KeyValuePair<int, float>));
    hipMalloc(&d_lo, sizeof(hipcub::KeyValuePair<int, float>));
    hipMalloc(&d_up, sizeof(hipcub::KeyValuePair<int, float>));

    // find axis extreme points
    findMax_cub<float>(d_ri, d_x, n);
    findMax_cub<float>(d_up, d_y, n);
    findMin_cub<float>(d_le, d_x, n);
    findMin_cub<float>(d_lo, d_y, n);


    // index corner points
    hipMalloc(&d_c1, sizeof(hipcub::KeyValuePair<int, float>));
    hipMalloc(&d_c2, sizeof(hipcub::KeyValuePair<int, float>));
    hipMalloc(&d_c3, sizeof(hipcub::KeyValuePair<int, float>));
    hipMalloc(&d_c4, sizeof(hipcub::KeyValuePair<int, float>));

    hipMalloc(&d_c, sizeof(float) * n);

    // compute the manhattan distance and find the minimum
    // corner 1
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, d_ri, d_up, n);
    hipDeviceSynchronize();
    findMin_cub<float>(d_c1, d_c, n);

    // corner 2
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, d_le, d_up, n);
    hipDeviceSynchronize();
    findMin_cub<float>(d_c2, d_c, n);

    // corner 3
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, d_le, d_lo, n);
    hipDeviceSynchronize();
    findMin_cub<float>(d_c3, d_c, n);

    // corner 4
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, d_ri, d_lo, n);
    hipDeviceSynchronize();
    findMin_cub<float>(d_c4, d_c, n);

    // slope device variables
    float *d_m1, *d_m2, *d_m3, *d_m4, *d_m1b, *d_m2b, *d_m3b, *d_m4b, *d_mh;
    hipMalloc(&d_m1, sizeof(float));
    hipMalloc(&d_m2, sizeof(float));
    hipMalloc(&d_m3, sizeof(float));
    hipMalloc(&d_m4, sizeof(float));
    hipMalloc(&d_m1b, sizeof(float));
    hipMalloc(&d_m2b, sizeof(float));
    hipMalloc(&d_m3b, sizeof(float));
    hipMalloc(&d_m4b, sizeof(float));
    hipMalloc(&d_mh, sizeof(float));

    // compute slopes
    gpu_compute_slopes<<<1,1>>>(d_m1, d_m2, d_m3, d_m4, d_m1b, d_m2b, d_m3b, d_m4b, d_mh,
                                d_ri, d_le, d_lo, d_up, d_ri, d_le, d_lo, d_up,
                                d_x, d_y, n);

    hipDeviceSynchronize();

    hipMalloc(&d_vec_inQ, sizeof(char) * n);
    hipMalloc(&d_qa, sizeof(INDEX) * n);
    hipMalloc(&d_q, sizeof(INDEX) * n);
    hipMalloc(&d_size, sizeof(INDEX));

    // is in polygon?
    kernel_inPointsInQ<char><<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_x, d_y, n, 
                d_ri, d_up, d_le, d_lo, d_c1, d_c2, d_c3, d_c4,
                d_m1, d_m2, d_m3, d_m4, d_m1b, d_m2b, d_m3b, d_m4b, d_mh, 
                d_vec_inQ, d_qa ); kernelCallCheck();
    hipDeviceSynchronize();

    compaction_cub<INDEX,char>(d_q, d_size, d_vec_inQ, d_qa, n);

    hipMemcpy(&size, d_size, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}





// copy the device variables to the host variables
void filter_cub_flagged::copy_to_host(){
   
   hipMalloc(&d_out_x, sizeof(float) * n);
   hipMalloc(&d_out_y, sizeof(float) * n);
   get_coor<char><<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_out_x, d_out_y, d_vec_inQ, d_q, d_x, d_y, d_size, n); kernelCallCheck();
   hipDeviceSynchronize();

   // output malloc
   hipMalloc(&out_x, sizeof(float) * n);
   hipMalloc(&out_y, sizeof(float) * n);

   // copy output to host
   hipMemcpy(out_x, d_out_x, sizeof(float) * n, hipMemcpyDeviceToHost);
   hipMemcpy(out_y, d_out_y, sizeof(float) * n, hipMemcpyDeviceToHost);//

}

// print indices and cooordinates of all axis extreme points, corners, and slopes
void filter_cub_flagged::print_extremes(){
    // copy from device to host
    hipMemcpy(&ri, d_ri, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&up, d_up, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&le, d_le, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&lo, d_lo, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&c1, d_c1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&c2, d_c2, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&c3, d_c3, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&c4, d_c4, sizeof(int), hipMemcpyDeviceToHost);

    xri = x[ri]; yri = y[ri];
    xle = x[le]; yle = y[le];
    xlo = x[lo]; ylo = y[lo];
    xup = x[up]; yup = y[up];
    xc1 = x[c1]; yc1 = y[c1];
    xc2 = x[c2]; yc2 = y[c2];
    xc3 = x[c3]; yc3 = y[c3];
    xc4 = x[c4]; yc4 = y[c4];

    printf("\n");
    printf("ri: %i, xri: %f, yri: %f\n", ri, xri, yri);
    printf("up: %i, xup: %f, yup: %f\n", up, xup, yup);
    printf("le: %i, xle: %f, yle: %f\n", le, xle, yle);
    printf("lo: %i, xlo: %f, ylo: %f\n", lo, xlo, ylo);
    printf("c1: %i, xc1: %f, yc1: %f\n", c1, xc1, yc1);
    printf("c2: %i, xc2: %f, yc2: %f\n", c2, xc2, yc2);
    printf("c3: %i, xc3: %f, yc3: %f\n", c3, xc3, yc3);
    printf("c4: %i, xc4: %f, yc4: %f\n", c4, xc4, yc4);
    //printf("m1: %f, m2: %f, m3: %f, m4: %f, mh: %f\n", m1, m2, m3, m4, mh);
    //printf("m1b: %f, m2b: %f, m3b: %f, m4b: %f\n", m1b, m2b, m3b, m4b);
    printf("\n");

    printf("compacted size: %d\n", size);

    // print x[q[i]] and y[q[i]] for all i in [0, size]
    for(int i = 0; i < size; i++)
        printf("-> %i: %f, %f\n", h_q[i], x[h_q[i]], y[h_q[i]]); //*/
}

void filter_cub_flagged::delete_filter(){
    // delete host variables
    //delete[] x;
    //delete[] y;
    delete h_q;

    hipDeviceSynchronize();
    kernelCallCheck();

    // delete device variables
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_ri);
    hipFree(d_le);
    hipFree(d_lo);
    hipFree(d_up);
    hipFree(d_c1);
    hipFree(d_c2);
    hipFree(d_c3);
    hipFree(d_c4);
    hipFree(d_c);
   /* hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_m3);
    hipFree(d_m4);
    hipFree(d_m1b);
    hipFree(d_m2b);
    hipFree(d_m3b);
    hipFree(d_m4b);
    hipFree(d_mh); // */
    hipFree(d_vec_inQ);
    hipFree(d_q);
    hipFree(d_qa);
    hipFree(d_size);

    hipDeviceSynchronize();
    kernelCallCheck();
}