// get cuda time for each kernel
//#include "cuda_time_m.cuh"

hipEvent_t start_event, stop_event;

cuda_time_m::cuda_time_m(){
    acc_time = 0;
    end_time = 0;
    time_acc = 0;
}

void cuda_time_m::init(){
    // start_event and stop_event are CUDA events
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipDeviceSynchronize();
}

void cuda_time_m::start(){
    hipEventRecord(start_event);
}

void cuda_time_m::pause(){
    hipEventRecord(stop_event);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&acc_time, start_event, stop_event);
    end_time += acc_time;
}

void cuda_time_m::stop(){
    hipEventRecord(stop_event);
    hipEventSynchronize(stop_event);
}

// get time 
float cuda_time_m::get_time(){
    return end_time;
}
