#include "hip/hip_runtime.h"
//#include "scan.cu"

// Compaction a sparce array of integers.
// The array is sparse and contains only a few elements.
// it used a auxiliary to determinate if a element is present in the array.
// The auxiliary is a array of integers.
// the output is a array of integers.
// The output array has the same size of the input array.
template <typename T, typename V>
void compaction_serial(T *output, INDEX *h_num, V *auxiliary, T *input, INDEX size){
    INDEX i, j=0;
    for (i = 0; i < size; i++)
    {
        if (auxiliary[i] == 1)
        {
            output[j] = input[i];
            j++;
        }
    }
    *h_num = j;
}

// Parallel compaction of a sparce array of integers.
// The array is sparse and contains only a few elements.
// it used a auxiliary to determinate if a element is present in the array.
// The auxiliary is a array of integers.
// the output is a array of integers.
// The output array has the same size of the input array.
template <typename T, typename V>
void compaction_parallel(T *output, INDEX *h_num, V *auxiliary, T *input, INDEX size){
    INDEX i, j=0;
    #pragma omp parallel for
    for (i = 0; i < size; i++)
    {
        if (auxiliary[i] == 1)
        {
            output[j] = input[i];
            j++;
        }
    }
    *h_num = j;
}

// store the elements of the input array using the auxiliary array.
// the auxiliary array said where the element is present in the input array.
// the input array is a array of integers.
// the auxiliary array is a array of integers.
// the output array is a array of integers.
// the output array has the same size of the input array.
// this function is in GPU.
template <typename T, typename V>
__global__ void store_array(T *output, INDEX *d_num, V *auxiliary, T *scan, T *input, INDEX n){
    INDEX off = threadIdx.x + blockIdx.x * blockDim.x;
    if (off < n){
        // debug: print of variable, the element of the input array, auxiliary array and output array.
        //printf("%d %d %d %d\n", off, input[off], auxiliary[off], output[off]);
        if (auxiliary[off] == 1){
            output[scan[off]] = input[off];
        }
        if (off == n-1)
            *d_num = scan[off] + auxiliary[off];
    }
}

// gpu compactation of a sparce array of integers.
// The array is sparse and contains only a few elements.
// it used a auxiliary to determinate if a element is present in the array.
// The auxiliary is a array of integers.
// the output is a array of integers.
// The output array has the same size of the input array.
template <typename T, typename V>
void compaction_cub_scan(T *d_out, INDEX *d_num, V *bit_vector, T *d_in, INDEX n){
    INDEX *aux_scan;
    hipMalloc(&aux_scan, sizeof(INDEX)*n);
    hipMemset(aux_scan, 0, sizeof(INDEX)*n);
    scan_parallel_cub<INDEX>(bit_vector, aux_scan, n);  kernelCallCheck();
    store_array<<<(n+BSIZE-1)/BSIZE, BSIZE>>>(d_out, d_num, bit_vector, aux_scan, d_in, n);
    hipDeviceSynchronize();  kernelCallCheck();
    //hipFree(aux_scan);
}

// compaction an array of T.
// using the cub library.
// the input array is a array of T.
// the output array is a array of T.
// the flag array is a array of V.
template <typename T, typename V>
void compaction_cub(T *d_out, INDEX *d_num_selected_out, V *d_flags, T *d_in, INDEX n){
    //char *d_flags;               // e.g., [1, 0, 0, 1, 0, 1, 1, 0]
    //int  *d_num_selected_out;    // e.g., [ ]
    //hipMalloc(&d_num_selected_out, sizeof(int));
    // Determine temporary device storage requirements
    void *d_temp_storage = nullptr;
    std::size_t temp_storage_bytes = 0;
    hipcub::DevicePartition::Flagged(d_temp_storage, temp_storage_bytes, d_in, d_flags, d_out, d_num_selected_out, n);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run selection
    hipcub::DevicePartition::Flagged(d_temp_storage, temp_storage_bytes, d_in, d_flags, d_out, d_num_selected_out, n);
    // Free temporary storage
    //hipFree(d_temp_storage);
}


template <typename T, typename B>
void compaction_scan_thrust(T *d_out, INDEX *d_num, B *d_bit_set, T *d_in, INDEX n){
    thrust::device_ptr<B> b_ptr = thrust::device_pointer_cast(d_bit_set);
	thrust::device_vector<T> q_out(n,0);
	thrust::exclusive_scan(thrust::device, b_ptr, b_ptr + n, q_out.begin(), 0);
	T *d_scan = thrust::raw_pointer_cast(&q_out[0]);
    store_array<<<(n+BSIZE-1)/BSIZE, BSIZE>>>(d_out, d_num, d_bit_set, d_scan, d_in, n);
    hipDeviceSynchronize();  kernelCallCheck();
    // Free temporary storage
    //hipFree(d_scan);
}

/*template <typename T, typename B>
void compaction_copy_thrust(T *d_in, B *d_bit_set, T *d_out, int n){
	auto result_end = thrust::copy_if(	thrust::device, p_ptr, p_ptr + n, q_vec.begin(), ff);
}*/

template <typename T, typename V>
__global__ void compact_partial_sums(T *output, INDEX *d_num, T *input, V *partial_sums, T *segmented_partial_sums, V *d_bit_vector, INDEX num_elements) {
	INDEX offset = threadIdx.x + blockIdx.x * blockDim.x;
	//const INDEX globalWarpIdx = (threadIdx.x + blockDim.x * blockIdx.x)/WARPSIZE;
	INDEX globalSegmentIdx = (threadIdx.x + blockDim.x * blockIdx.x)/WMMA_TILE_SIZE;
	if (offset < num_elements) {
        INDEX ind = (INDEX) partial_sums[offset] + segmented_partial_sums[globalSegmentIdx];
        if ((int)d_bit_vector[offset] == 1){
            if (ind > 0)
                output[ind-1] = input[offset];
        }
        if (offset == num_elements - 1)
            *d_num = ind;
	}
}


template <typename T, typename V>
void compaction_tc_scan(T *d_out, INDEX *d_num, V *d_bit_vector, T *d_in, INDEX n){
	INDEX num_segments = (n + WMMA_TILE_SIZE - 1)/WMMA_TILE_SIZE + 1; //(n + SEGMENT_SIZE - 1) / SEGMENT_SIZE;
    dim3 blockDim(BLOCK_DIM,1,1);
    dim3 gridDim((n + 8192 - 1)/8192,1,1);
	T *segmented_partial_sums;
	V *partial_sums; kernelCallCheck();
	hipMalloc(&partial_sums,sizeof(V)*n); kernelCallCheck();
	hipMalloc(&segmented_partial_sums,sizeof(T)*num_segments); kernelCallCheck();
    compute_wmma_segmented_prefixsum_256n_block_ps<T,V><<<gridDim, blockDim>>>(d_bit_vector, partial_sums, segmented_partial_sums, n); kernelCallCheck();
    hipDeviceSynchronize();
	scan_parallel_cub<T>(segmented_partial_sums,segmented_partial_sums,num_segments); kernelCallCheck();
	compact_partial_sums<T,V><<<(n+BSIZE-1)/BSIZE+1, BSIZE>>>(d_out, d_num, d_in, partial_sums, segmented_partial_sums, d_bit_vector, n); kernelCallCheck();
    hipDeviceSynchronize();//*/
	// free memorry
	hipFree(segmented_partial_sums);
	hipFree(partial_sums);
}