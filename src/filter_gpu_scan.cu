//#include "filter_gpu_scan.cuh"
#include "kernel/kernel_gpu_scan.cu"

// compacting vector
half *d_vec_inQ;

filter_gpu_scan::filter_gpu_scan(float *x_in, float *y_in, INDEX siz2e){
    x = x_in;
    y = y_in;
    n = size2;
    gpu_scan();
    //f_gpu_scan();

    // save the time for deleting
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // cuda copy d_q to host
    h_q = new INDEX[size];
    hipMemcpy(h_q, d_q, sizeof(INDEX) * size, hipMemcpyDeviceToHost); kernelCallCheck();
    //print_extremes();

    // save the time for copying to host
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_copy2host = (milliseconds - t_copy2host) / step + t_copy2host;
}

void filter_gpu_scan::gpu_scan(){
    // GET CUDA TIME
    hipDeviceSynchronize();
    step++;
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // cuda malloc for x and y arrays
    hipMalloc(&d_x, sizeof(float) * n);
    hipMalloc(&d_y, sizeof(float) * n);
    
    // cuda mem copy for x and y arrays
    hipMemcpy(d_x, x, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc(&d_ri, sizeof(INDEX));
    hipMalloc(&d_le, sizeof(INDEX));
    hipMalloc(&d_lo, sizeof(INDEX));
    hipMalloc(&d_up, sizeof(INDEX));

    // save the time for copying to device
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_copy2device = (milliseconds - t_copy2device) / step + t_copy2device;

    hipEvent_t start_filter, stop_filter;
    hipEventCreate(&start_filter);
    hipEventCreate(&stop_filter);
    hipEventRecord(start_filter);

    // get the time for finding axis extreme points
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    // find axis extreme points
    findMax_kernel<float>(d_ri, d_x, n); kernelCallCheck();
    findMax_kernel<float>(d_up, d_y, n); kernelCallCheck();
    findMin_kernel<float>(d_le, d_x, n); kernelCallCheck();
    findMin_kernel<float>(d_lo, d_y, n); kernelCallCheck();

    // copy from d_ri to host
    hipMemcpy(&ri, d_ri, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&up, d_up, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&le, d_le, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&lo, d_lo, sizeof(INDEX), hipMemcpyDeviceToHost);

    xri = x[ri]; yri = y[ri];
    xle = x[le]; yle = y[le];
    xlo = x[lo]; ylo = y[lo];
    xup = x[up]; yup = y[up];

    // save the time for finding axis extreme points
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_find_extremes = (milliseconds - t_find_extremes) / step + t_find_extremes;

    // get the time for finding corner points
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipMalloc(&d_c1, sizeof(INDEX));
    hipMalloc(&d_c2, sizeof(INDEX));
    hipMalloc(&d_c3, sizeof(INDEX));
    hipMalloc(&d_c4, sizeof(INDEX));

    hipMalloc(&d_c, sizeof(float) * n);
    
    // compute the manhattan distance and find the minimum
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, xri, yup, n);
    hipDeviceSynchronize();
    findMin_kernel<float>(d_c1, d_c, n);
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, xle, yup, n);
    hipDeviceSynchronize();
    findMin_kernel<float>(d_c2, d_c, n);
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, xle, ylo, n);
    hipDeviceSynchronize();
    findMin_kernel<float>(d_c3, d_c, n);
    compute_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c, d_x, d_y, xri, ylo, n);
    hipDeviceSynchronize();
    findMin_kernel<float>(d_c4, d_c, n);

    /*// find corner points
    findMin_kernel_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c1, d_c, d_x, d_y, xri, yup, n); kernelCallCheck();
    findMin_kernel_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c2, d_c, d_x, d_y, xle, yup, n); kernelCallCheck();
    findMin_kernel_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c3, d_c, d_x, d_y, xle, ylo, n); kernelCallCheck();
    findMin_kernel_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c4, d_c, d_x, d_y, xri, ylo, n); kernelCallCheck();*/

    // copy from d_c1 to host
    hipMemcpy(&c1, d_c1, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&c2, d_c2, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&c3, d_c3, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&c4, d_c4, sizeof(INDEX), hipMemcpyDeviceToHost);

    xc1 = x[c1]; yc1 = y[c1];
    xc2 = x[c2]; yc2 = y[c2];
    xc3 = x[c3]; yc3 = y[c3];
    xc4 = x[c4]; yc4 = y[c4];


    // save the time for finding corner points
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_find_corners = (milliseconds - t_find_corners) / step + t_find_corners;

    // get the time for finding points in q
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    computeSlopes();

    hipMalloc(&d_vec_inQ, sizeof(half) * n); kernelCallCheck();
    hipMalloc(&d_qa, sizeof(INDEX) * n); kernelCallCheck();
    hipMalloc(&d_q, sizeof(INDEX) * n); kernelCallCheck();
    hipMalloc(&d_size, sizeof(INDEX)); kernelCallCheck();

    // is in polygon?
    kernel_inPointsInQ<half><<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_x, d_y, n, 
                xri, yri, xup, yup, xle, yle, xlo, ylo,
                xc1, yc1, xc2, yc2, xc3, yc3, xc4, yc4,
                m1, m2, m3, m4, mh, m1b, m2b, m3b, m4b, 
                d_vec_inQ, d_qa ); kernelCallCheck();
    hipDeviceSynchronize();

    /*// copy d_vec_inQ to host and print all element of h_vec_inQ
    half *h_vec_inQ = new half[n];
    hipMemcpy(h_vec_inQ, d_vec_inQ, sizeof(half) * n, hipMemcpyDeviceToHost); kernelCallCheck();
    INDEX *h_qa = new INDEX[n];
    hipMemcpy(h_qa, d_qa, sizeof(INDEX) * n, hipMemcpyDeviceToHost); kernelCallCheck();
    for (int i = 0; i < n; i++){
        if ((int)h_vec_inQ[i] == 1){
            printf("%d %i\n", i, (int)h_qa[i]);
        }
        //printf("%i %i\n", i, (int) h_vec_inQ[i]);
    } //*/

    // save the time for finding points in q
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_find_points_in_Q = (milliseconds - t_find_points_in_Q) / step + t_find_points_in_Q;

    // get the time for compacting 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    compaction_tc_scan<INDEX,half>(d_q, d_size, d_vec_inQ, d_qa, n);
    //hipDeviceSynchronize();
    hipMemcpy(&size, d_size, sizeof(INDEX), hipMemcpyDeviceToHost);

    // save the time for compacting
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);    
    t_compaction = (milliseconds - t_compaction) / step + t_compaction;

    /*// copy d_qa to host and print all element
    INDEX *h_qa = new INDEX[n];
    hipMemcpy(h_qa, d_q, sizeof(INDEX) * n, hipMemcpyDeviceToHost); kernelCallCheck();
    for (int i = 0; i < size; i++){
        printf("%i %i\n", i,(int) h_qa[i]);
    }*/

    // save the time for compacting
    hipEventRecord(stop_filter);
    hipEventSynchronize(stop_filter);
    hipEventElapsedTime(&milliseconds, start_filter, stop_filter);    
    t_total = (milliseconds - t_total) / step + t_total;
}



void filter_gpu_scan::f_gpu_scan(){
    // cuda malloc for x and y arrays
    hipMalloc(&d_x, sizeof(float) * n);
    hipMalloc(&d_y, sizeof(float) * n);

    // cuda mem copy for x and y arrays
    hipMemcpy(d_x, x, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc(&d_ri, sizeof(INDEX));
    hipMalloc(&d_le, sizeof(INDEX));
    hipMalloc(&d_lo, sizeof(INDEX));
    hipMalloc(&d_up, sizeof(INDEX));

    // find axis extreme points
    findMax_kernel<float>(d_ri, d_x, n); kernelCallCheck();
    findMax_kernel<float>(d_up, d_y, n); kernelCallCheck();
    findMin_kernel<float>(d_le, d_x, n); kernelCallCheck();
    findMin_kernel<float>(d_lo, d_y, n); kernelCallCheck();


    hipMalloc(&d_c1, sizeof(INDEX));
    hipMalloc(&d_c2, sizeof(INDEX));
    hipMalloc(&d_c3, sizeof(INDEX));
    hipMalloc(&d_c4, sizeof(INDEX));

    hipMalloc(&d_c, sizeof(float) * n);

    // find corner points
    findMin_kernel_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c1, d_c, d_x, d_y, d_ri, d_up, n); kernelCallCheck();
    findMin_kernel_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c2, d_c, d_x, d_y, d_le, d_up, n); kernelCallCheck();
    findMin_kernel_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c3, d_c, d_x, d_y, d_le, d_lo, n); kernelCallCheck();
    findMin_kernel_manhattan<<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_c4, d_c, d_x, d_y, d_ri, d_lo, n); kernelCallCheck();

    hipMalloc(&d_m1, sizeof(float));
    hipMalloc(&d_m2, sizeof(float));
    hipMalloc(&d_m3, sizeof(float));
    hipMalloc(&d_m4, sizeof(float));
    hipMalloc(&d_m1b, sizeof(float));
    hipMalloc(&d_m2b, sizeof(float));
    hipMalloc(&d_m3b, sizeof(float));
    hipMalloc(&d_m4b, sizeof(float));
    hipMalloc(&d_mh, sizeof(float));

    // compute slopes
    gpu_compute_slopes<<<1,1>>>(d_m1, d_m2, d_m3, d_m4, d_m1b, d_m2b, d_m3b, d_m4b, d_mh,
                                d_ri, d_le, d_lo, d_up, d_c1, d_c2, d_c3, d_c4,
                                d_x, d_y, n); kernelCallCheck();
    hipDeviceSynchronize();

    hipMalloc(&d_vec_inQ, sizeof(half) * n);
    hipMalloc(&d_qa, sizeof(INDEX) * n);
    hipMalloc(&d_q, sizeof(INDEX) * n);
    hipMalloc(&d_size, sizeof(INDEX));

    // is in polygon?
    kernel_inPointsInQ<half><<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_x, d_y, n, 
                d_ri, d_up, d_le, d_lo, d_c1, d_c2, d_c3, d_c4,
                d_m1, d_m2, d_m3, d_m4, d_m1b, d_m2b, d_m3b, d_m4b, d_mh, 
                d_vec_inQ, d_qa ); kernelCallCheck();
    hipDeviceSynchronize();

    compaction_tc_scan<INDEX,half>(d_q, d_size, d_vec_inQ, d_qa, n);

    hipMemcpy(&size, d_size, sizeof(INDEX), hipMemcpyDeviceToHost);

}




// copy the device variables to the host variables
void filter_gpu_scan::copy_to_host(){
   
   hipMalloc(&d_out_x, sizeof(float) * n);
   hipMalloc(&d_out_y, sizeof(float) * n);
   get_coor<half><<<(n+BSIZE-1)/BSIZE,BSIZE>>>(d_out_x, d_out_y, d_vec_inQ, d_q, d_x, d_y, d_size, n); kernelCallCheck();
   hipDeviceSynchronize();

   // output malloc
   hipMalloc(&out_x, sizeof(float) * n);
   hipMalloc(&out_y, sizeof(float) * n);

   // copy output to host
   hipMemcpy(out_x, d_out_x, sizeof(float) * n, hipMemcpyDeviceToHost);
   hipMemcpy(out_y, d_out_y, sizeof(float) * n, hipMemcpyDeviceToHost);//

}


void filter_gpu_scan::delete_filter(){
    // get the time for deleting
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    // delete host variables
    //delete[] x;
    //delete[] y;
    delete h_q;

    hipDeviceSynchronize();
    kernelCallCheck();

    // delete device variables
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_ri);
    hipFree(d_le);
    hipFree(d_lo);
    hipFree(d_up);
    hipFree(d_c1);
    hipFree(d_c2);
    hipFree(d_c3);
    hipFree(d_c4);
    hipFree(d_c);
   /* hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_m3);
    hipFree(d_m4);
    hipFree(d_m1b);
    hipFree(d_m2b);
    hipFree(d_m3b);
    hipFree(d_m4b);
    hipFree(d_mh); // */
    hipFree(d_vec_inQ);
    hipFree(d_q);
    hipFree(d_qa);
    hipFree(d_size);

    hipDeviceSynchronize();
    kernelCallCheck();

    // save the time for deleting
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    t_delete = (milliseconds - t_delete) / step;
}

// print indices and cooordinates of all axis extreme points, corners, and slopes
void filter_gpu_scan::print_extremes(){
    // copy from device to host
    hipMemcpy(&ri, d_ri, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&up, d_up, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&le, d_le, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&lo, d_lo, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&c1, d_c1, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&c2, d_c2, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&c3, d_c3, sizeof(INDEX), hipMemcpyDeviceToHost);
    hipMemcpy(&c4, d_c4, sizeof(INDEX), hipMemcpyDeviceToHost);

    xri = x[ri]; yri = y[ri];
    xle = x[le]; yle = y[le];
    xlo = x[lo]; ylo = y[lo];
    xup = x[up]; yup = y[up];
    xc1 = x[c1]; yc1 = y[c1];
    xc2 = x[c2]; yc2 = y[c2];
    xc3 = x[c3]; yc3 = y[c3];
    xc4 = x[c4]; yc4 = y[c4];

    printf("\n");
    printf("ri: %i, xri: %f, yri: %f\n", (INDEX)ri, xri, yri);
    printf("up: %i, xup: %f, yup: %f\n", (INDEX)up, xup, yup);
    printf("le: %i, xle: %f, yle: %f\n", (INDEX)le, xle, yle);
    printf("lo: %i, xlo: %f, ylo: %f\n", (INDEX)lo, xlo, ylo);
    printf("c1: %i, xc1: %f, yc1: %f\n", (INDEX)c1, xc1, yc1);
    printf("c2: %i, xc2: %f, yc2: %f\n", (INDEX)c2, xc2, yc2);
    printf("c3: %i, xc3: %f, yc3: %f\n", (INDEX)c3, xc3, yc3);
    printf("c4: %i, xc4: %f, yc4: %f\n", (INDEX)c4, xc4, yc4);
    //printf("m1: %f, m2: %f, m3: %f, m4: %f, mh: %f\n", m1, m2, m3, m4, mh);
    //printf("m1b: %f, m2b: %f, m3b: %f, m4b: %f\n", m1b, m2b, m3b, m4b);
    printf("\n");

    printf("compacted size: %d\n", size);

    // print x[q[i]] and y[q[i]] for all i in [0, size]
    for(int i = 0; i < size; i++)
        printf("-> %i: %f, %f\n", h_q[i], x[h_q[i]], y[h_q[i]]); //*/
}